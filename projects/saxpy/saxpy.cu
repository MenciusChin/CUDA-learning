#include "hip/hip_runtime.h"
/*
SAXPY example By Mark Harris
https://developer.nvidia.com/blog/easy-introduction-cuda-c-and-c/
*/

#include <stdio.h>

__global__ void saxpy(int n, float a, float *x, float *y)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < n) y[i] = a*x[i] + y[i];
}


int main(void)
{
    // Initialize variables
    int N = 1<<20;
    float *x, *y, *d_x, *d_y;

    // Allocate host memories
    x = (float*)malloc(N*sizeof(float));
    y = (float*)malloc(N*sizeof(float));

    // Allocate device memories
    hipMalloc(&d_x, N*sizeof(float)); 
    hipMalloc(&d_y, N*sizeof(float));

    // Initialize host arrays
    for (int i = 0; i < N; i++) {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    // Initialize device arrays
    hipMemcpy(d_x, x, N*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, N*sizeof(float), hipMemcpyHostToDevice);

    // Perform SAXPY on 1M elements
    saxpy<<<(N+255)/256, 256>>>(N, 2.0f, d_x, d_y);

    // Retrive result back to host
    hipMemcpy(y, d_y, N*sizeof(float), hipMemcpyDeviceToHost);

    float maxError = 0.0f;
    for (int i = 0; i < N; i++)
        maxError = max(maxError, abs(y[i]-4.0f));
    printf("Max error: %f\n", maxError);

    // Cleaning up memories
    hipFree(d_x);
    hipFree(d_y);
    free(x);
    free(y);
}